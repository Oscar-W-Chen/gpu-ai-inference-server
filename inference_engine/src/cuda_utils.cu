#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <hip/hip_runtime.h>
#include <iostream>

namespace inference{
namespace cuda{

// CUDA kernel for vector addition
// This runs on the GPU - each thread handles one element
__global__ void addVectors(const float* a, const float* b, float* result, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

bool IsCudaAvailable(){
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess){
        std::cerr << "CUDA error checking for devices: "
                  << hipGetErrorString(error) << std::endl;
        return false;  
    }

    return deviceCount > 0;
}

int GetDeviceCount(){
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess){
        std::cerr << "CUDA error getting device count: "
                  << hipGetErrorString(error) << std::endl;
        return 0;
    }
    return deviceCount;
}

std::string GetDeviceInfo(int device_id) {
    hipDeviceProp_t deviceProp;
    hipError_t error = hipGetDeviceProperties(&deviceProp, device_id);
    if (error != hipSuccess){
        std::cerr << "CUDA error getting device properties: "
                  << hipGetErrorString(error) << std::endl;
        return "Unknown device";
    }

    // Format device information
    std::string info =  "Device " + std::to_string(device_id) + ": " + deviceProp.name +
                        " (Compute Capability " +
                        std::to_string(deviceProp.major) + "." +
                        std::to_string(deviceProp.minor) + ")";
    
    return info;
}

__host__ bool VectorAdd(const std::vector<float>& a, const std::vector<float>& b, std::vector<float>& result){
    // Validate input sizes
    if(a.size() != b.size()) {
        std::cerr << "Vector sizes does not match" << std::endl;
        return false;
    }
    int size = a.size();
    result.resize(size);

    // Allocate device memory
    float *d_a = nullptr, *d_b = nullptr, *d_result = nullptr;

    // Helper function to clean up resources
    auto cleanup = [&]() {
        if (d_a) hipFree(d_a);
        if (d_b) hipFree(d_b);
        if (d_result) hipFree(d_result);
    }

    // Allocate memory on GPU for first input vector
    hipError_t error = hipMalloc(&d_a, size * sizeof(float));
    if (error != hipSuccess) {
        std::cerr  << "Failed to allocate device memory for vector A: " 
                   << hipGetErrorString(error) << std::endl;
        cleanup();
        return false;
    }

    // Allocate memory on GPU for second input vector
    error = hipMalloc(&d_b, size * sizeof(float));
    if (error != hipSuccess) {
        std::cerr  << "Failed to allocate device memory for vector B: " 
                   << hipGetErrorString(error) << std::endl;
        cleanup();
        return false;
    }

    // Allocate memory on GPU for result vector
    error = hipMalloc(&d_result, size * sizeof(float));
    if (error != hipSuccess) {
        std::cerr  << "Failed to allocate device memory for vector result: " 
                   << hipGetErrorString(error) << std::endl;
        cleanup();
        return false;
    }

    // Copy input data from host to device (CPU to GPU)
    error = hipMemcpy(d_a, a.data(), size * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        std::cerr << "Failed to copy vector A to device: "
                  << hipGetErrorString(error) << std::endl;
        cleanup();
        return false;
    }
    
    error = hipMemcpy(d_b, b.data(), size * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        std::cerr << "Failed to copy vector B to device: "
                  << hipGetErrorString(error) << std::endl;
        cleanup();
        return false;
    }

    // Configure kernel execution parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector addition kernel on GPU
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);

    // Wait for GPU to finish
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr <<< "Kernel execution failed: "
                  <<< hipGetErrorString(error) <<< std::endl;
        cleanup();
        return false;
    }

    // Copy the result back to host (GPU to CPU)
    error = hipMemcpy(result.data(), d_result, size * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        std::cerr <<< "Failed to copy result vector from device: "
                  <<< hipGetErrorString(error) <<< std::endl;
        cleanup();
        return false;
    }

    // Free device memory
    cleanup();
    return true;
}

MemoryInfo GetMemoryInfo(int device_id){
    MemoryInfo memInfo = {0, 0, 0};

    // Set device
    hipError_t error = hipSetDevice(device_id);
    if (error != hipSuccess) {
        std::cerr << "Failed to set device: " 
                  << hipGetErrorString(error) << std::endl;
        return memInfo;
    }

    // Get memory info
    size_t free, total;
    error = hipMemGetInfo(&free, &total);
    if (error != hipSuccess) {
        std::cerr << "Failed to get memory info: " 
                  << hipGetErrorString(error) << std::endl;
        return memInfo;
    }

    memInfo.free = free;
    memInfo.total = total;
    memInfo.used = total - free;
    return memInfo;
}


}
}